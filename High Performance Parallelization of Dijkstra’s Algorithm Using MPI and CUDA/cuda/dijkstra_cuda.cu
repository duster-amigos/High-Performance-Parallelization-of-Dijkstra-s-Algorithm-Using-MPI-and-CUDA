#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <limits>
#include <hip/hip_runtime.h>
#include <chrono>
#include <string>
#include <getopt.h>

#define INF 1000000000

__global__ void dijkstra_kernel(int* adj, int* dist, int* pred, bool* updated, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n) return;
    for (int v = 0; v < n; ++v) {
        int weight = adj[tid * n + v];
        if (weight != INF && dist[tid] != INF) {
            int new_dist = dist[tid] + weight;
            int old_dist = atomicMin(&dist[v], new_dist);
            if (new_dist < old_dist) {
                pred[v] = tid;
                updated[v] = true;
            }
        }
    }
}

void print_usage() {
    std::cout << "Usage: dijkstra_cuda -i <adj_matrix> -s <source> -n <nodes>\n";
}

int main(int argc, char* argv[]) {
    int n = 0, source = 0;
    std::string input_file;
    int opt;
    while ((opt = getopt(argc, argv, "i:s:n:")) != -1) {
        switch (opt) {
            case 'i': input_file = optarg; break;
            case 's': source = std::stoi(optarg); break;
            case 'n': n = std::stoi(optarg); break;
            default: print_usage(); return 1;
        }
    }
    if (n <= 0 || input_file.empty() || source < 0 || source >= n) {
        print_usage();
        return 1;
    }
    try {
        std::vector<int> h_adj(n * n);
        std::ifstream ifs(input_file);
        if (!ifs) throw std::runtime_error("Cannot open input file");
        for (int i = 0; i < n * n; ++i) {
            int val; ifs >> val;
            h_adj[i] = (val == -1 ? INF : val);
        }
        ifs.close();
        std::vector<int> h_dist(n, INF);
        std::vector<int> h_pred(n, -1);
        std::vector<bool> h_updated(n, false);
        h_dist[source] = 0;
        int *d_adj, *d_dist, *d_pred;
        bool *d_updated;
        hipMalloc(&d_adj, n * n * sizeof(int));
        hipMalloc(&d_dist, n * sizeof(int));
        hipMalloc(&d_pred, n * sizeof(int));
        hipMalloc(&d_updated, n * sizeof(bool));
        hipMemcpy(d_adj, h_adj.data(), n * n * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_dist, h_dist.data(), n * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_pred, h_pred.data(), n * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_updated, h_updated.data(), n * sizeof(bool), hipMemcpyHostToDevice);
        auto start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < n; ++i) {
            dijkstra_kernel<<<(n+255)/256, 256>>>(d_adj, d_dist, d_pred, d_updated, n);
            hipDeviceSynchronize();
        }
        hipMemcpy(h_dist.data(), d_dist, n * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_pred.data(), d_pred, n * sizeof(int), hipMemcpyDeviceToHost);
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end - start;
        std::cout << "Dijkstra (CUDA) completed in " << elapsed.count() << " seconds.\n";
        std::cout << "Distances from source " << source << ": ";
        for (int d : h_dist) std::cout << (d == INF ? -1 : d) << " ";
        std::cout << "\nPredecessors: ";
        for (int p : h_pred) std::cout << p << " ";
        std::cout << std::endl;
        hipFree(d_adj); hipFree(d_dist); hipFree(d_pred); hipFree(d_updated);
    } catch (const std::exception& ex) {
        std::cerr << "Error: " << ex.what() << std::endl;
        return 1;
    }
    return 0;
} 